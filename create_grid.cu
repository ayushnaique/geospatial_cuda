#include <bits/stdc++.h>
#include <hip/hip_runtime.h>

#include <cmath>
#include <fstream>
#include <sstream>
#include <vector>
using namespace std;

struct Point {
	int x, y;

	Point(int xc, int yc) : x(xc), y(yc) {}
};

struct Grid {
    Point *bottom_left, *bottom_right, *top_left, *top_right;

    Grid(Point* bl, Point* br, Point* tl, Point* tr)
                : bottom_left(bl), bottom_right(br), top_left(tl), top_right(tr) {}
};

__global__ void categorize_points(Point *d_points, int *d_categories, int *grid_counts, int count,
								  int range, int middle) {
    extern __shared__ int subgrid_counts[];

	int start = ((blockIdx.x * blockDim.x) + threadIdx.x) * range;
    if (threadIdx.x == 0) {
        subgrid_counts[0] = 0;
        subgrid_counts[1] = 0;
        subgrid_counts[2] = 0;
        subgrid_counts[3] = 0;
    }
    __syncthreads();

    int first = 0, second = 0, third = 0, fourth = 0;
	for (int i = start; i < start + range; i++) {
		if (i < count) {
            if (d_points[i].x <= middle and d_points[i].y <= middle){
				d_categories[i] = 1;
                first++;
            }
            else if (d_points[i].x > middle and d_points[i].y <= middle){
				d_categories[i] = 2;
                second++;
            }
            else if (d_points[i].x <= middle and d_points[i].y > middle){
				d_categories[i] = 3;
                third++;
            }
            else if (d_points[i].x > middle and d_points[i].y > middle){
				d_categories[i] = 4;
                fourth++;
            }
		}
	}
    atomicAdd(&subgrid_counts[0], first);
    atomicAdd(&subgrid_counts[1], second);
    atomicAdd(&subgrid_counts[2], third);
    atomicAdd(&subgrid_counts[3], fourth);
    __syncthreads();


    if (threadIdx.x == 0) {
        atomicAdd(&grid_counts[0], subgrid_counts[0]);
        atomicAdd(&grid_counts[1], subgrid_counts[1]);
        atomicAdd(&grid_counts[2], subgrid_counts[2]);
        atomicAdd(&grid_counts[3], subgrid_counts[3]);
    }
}

void quadtree_grid(vector<Point> points, int count, int dimension) {
	Point *d_points;

	int *d_categories, *d_grid_counts;
    vector<int> h_categories(count);
    vector<int> h_grid_counts(4);

	hipMalloc(&d_points, count * sizeof(Point));
	hipMalloc(&d_categories, count * sizeof(int));
	hipMalloc(&d_grid_counts, 4 * sizeof(int));

	hipMemcpy(d_points, points.data(), count * sizeof(Point),
			   hipMemcpyHostToDevice);

	int range, num_blocks = 16, threads_per_block = 256;
	if (count < num_blocks * threads_per_block)
		range = 1;
	else if (count % (num_blocks * threads_per_block) == 0)
		range = count / (threads_per_block * num_blocks);
	else {
		float value = static_cast<float>(count) / (num_blocks * threads_per_block);
		range = std::ceil(value);
	}
	printf("GPU: %d blocks of %d threads each with range=%d\n", num_blocks,
		   threads_per_block, range);

	dim3 grid(num_blocks, 1, 1);
	dim3 block(threads_per_block, 1, 1);
	categorize_points<<<grid, block, 4 * sizeof(int)>>>(d_points, d_categories, d_grid_counts, count, range,
									   dimension / 2);

	hipMemcpy(h_categories.data(), d_categories, count * sizeof(int),
			   hipMemcpyDeviceToHost);
	hipMemcpy(h_grid_counts.data(), d_grid_counts, 4 * sizeof(int),
			   hipMemcpyDeviceToHost);

    hipFree(d_points);
    hipFree(d_categories);
    hipFree(d_grid_counts);

    //for(int i = 0; i<1000; i++){
        //printf("x = %d, y = %d, category = %d\n", points[i].x, points[i].y, h_categories[i]);
    //}
    //int total = 0;
    //for(int i = 0; i<4; i++){
        //printf("sub grid %d - %d\n", i+1, h_grid_counts[i]);
        //total += h_grid_counts[i];
    //}
    //printf("Total Count - %d\n", count);
    //if(total == count){
        //printf("Matches\n");
    //}
}

int main() {
	string filename = "points.txt";
	vector<Point> points;
	int point_count = 0;

	ifstream file(filename);
	if (!file) {
		cerr << "Error: Could not open the file " << filename << endl;
		return 1;
	}

	string line;
	int x, y;

	while (getline(file, line)) {
		istringstream iss(line);
		if (iss >> x >> y) {
			Point p = Point(x, y);
			points.emplace_back(p);
			point_count++;
		} else {
			cerr << "Warning: Skipping malformed line: " << line << endl;
		}
	}

	file.close();

	quadtree_grid(points, point_count, 1000);

	return 0;
}
